#include "hip/hip_runtime.h"
#include "uammd.cuh"

#include "UAMMDstructured.cuh"

#include "Integrator/VerletNVE.cuh"

using Units = uammd::structured::UnitsSystem::KCALMOL_A;

using ffGeneric = uammd::structured::forceField::Generic::Generic<Units,
                                                                  uammd::structured::Types::BASIC,
                                                                  uammd::structured::conditions::excludedIntraInterChargedInter>;

int main(int argc, char** argv){

    auto sys = std::make_shared<uammd::System>();
    
    ullint seed = 0xf31337Bada55D00dULL^time(NULL);
    sys->rng().setSeed(seed);

    uammd::InputFile in(argv[1]);

    auto pd = uammd::structured::Wrapper::setUpParticleData(sys,in);
    auto pg = uammd::structured::Wrapper::setUpParticleGroup(pd,in);
    auto ff = uammd::structured::Wrapper::setUpForceField<ffGeneric>(pg,in);
  
    typename uammd::VerletNVE::Parameters par;

    in.getOption("dt",uammd::InputFile::Required)>>par.dt;

    par.dt     = par.dt*Units::TO_INTERNAL_TIME;
    par.initVelocities = false;

    uammd::real T;
    in.getOption("T",uammd::InputFile::Required)>>T;

    uammd::real kBT = Units::KBOLTZ*T;
    uammd::System::log<uammd::System::MESSAGE>("[VelocityGeneration] kBT:%f",kBT);

    uammd::structured::IntegratorBasic_ns::generateVelocity(pg,kBT,"VelocityGeneration");

    auto integrator = std::make_shared<uammd::VerletNVE>(pd, par);
    
    integrator->addInteractor(ff);

    int nSteps, nStepsInfoInterval, nStepsWriteInterval, nStepsMeasureInterval;
    
    in.getOption("nSteps",uammd::InputFile::Required)>>nSteps;
    in.getOption("nStepsInfoInterval",uammd::InputFile::Required)>>nStepsInfoInterval;
    in.getOption("nStepsWriteInterval",uammd::InputFile::Required)>>nStepsWriteInterval;
    in.getOption("nStepsMeasureInterval",uammd::InputFile::Required)>>nStepsMeasureInterval;
    
    uammd::structured::WriteStep<Units>::Parameters wParam = uammd::structured::WriteStep<Units>::inputFileToParam(in);
    
    std::shared_ptr<uammd::structured::WriteStep<Units>> wStep = std::make_shared<uammd::structured::WriteStep<Units>>(pg,
                                                                                                                       nStepsWriteInterval,
                                                                                                                       wParam);
    
    std::shared_ptr<uammd::structured::EnergyMeasure<ffGeneric>> eStep = std::make_shared<uammd::structured::EnergyMeasure<ffGeneric>>(pg,
                                                                                                                                       nStepsMeasureInterval,
                                                                                                                                       "energy.dat",
                                                                                                                                       ff);
    
    std::shared_ptr<uammd::structured::InertiaMeasure> iStep = std::make_shared<uammd::structured::InertiaMeasure>(pg,
                                                                                                                   nStepsMeasureInterval,
                                                                                                                   "inertia.dat");

    wStep->tryInit();
    eStep->tryInit();
    iStep->tryInit();

    wStep->tryApplyStep(0,true);
    eStep->tryApplyStep(0,true);
    iStep->tryApplyStep(0,true);
  
    uammd::Timer tim;
    tim.tic();
    forj(0, nSteps){
        integrator->forwardTime();
        if(nStepsInfoInterval > 0 and j%nStepsInfoInterval==0){
            uammd::System::log<uammd::System::MESSAGE>("Current step: %i",j);
        }
        if(nStepsWriteInterval > 0 and j%nStepsWriteInterval==0){
            wStep->tryApplyStep(j);
        }
        if(nStepsMeasureInterval > 0 and j%nStepsMeasureInterval==0){
            eStep->tryApplyStep(j);
            iStep->tryApplyStep(j);
        }
    }

    auto totalTime = tim.toc();
    uammd::System::log<uammd::System::MESSAGE>("mean FPS: %.2f", nSteps/totalTime);
    sys->finish();
    
    return EXIT_SUCCESS;
}
